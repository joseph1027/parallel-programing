#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include <WINDOWS.H>
#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<time.h>

// includes, cuda
#include <hip/hip_vector_types.h>
#include <driver_functions.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// CUDA utilities and system includes
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <math.h>
using namespace std;
using namespace cv;

int image[200000000];
int f_image[200000000];

/*__global__ void gradient_x(int x, int y,int width)
{
int gx;
gx = image[(x - 1)*width + (y - 1)] * 1 +
image[(x - 1)*width + y] * 2 +
image[(x - 1)*width + (y + 1)] * 1 +
image[(x + 1)*width + (y - 1)] * (-1) +
image[(x + 1)*width + y] * (-2) +
image[(x + 1)*width + (y + 1)] * (-1);
}
__global__ void gradient_y(int x, int y,int width)
{
int gy;
gy = image[(x - 1)*width + (y - 1)] * 1 +
image[x*width + (y - 1)] * 2 +
image[(x + 1)*width + (y - 1)] * 1 +
image[(x - 1)*width + (y + 1)] * (-1) +
image[x*width + (y + 1)] * (-2) +
image[(x + 1)*width + (y + 1)] * (-1);
}*/

__global__ void gradient(int* image, int* f_image, int length, int width)
{
	//int a = gradient_x(x, y,width);
	//int b = gradient_y(x, y,width);
	int templates[25] = { 
		1, 4, 7, 4, 1,

		4, 16, 26, 16, 4,

		7, 26, 41, 26, 7,

		4, 16, 26, 16, 4,

		1, 4, 7, 4, 1 };

	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id >= length*width)
	{
		return;
	}

	int y = id / width;
	int x = id % width;
	if (y < 2 || y >= length - 2 || x < 2 || x >= width - 2)
	{
		return;
	}
	/*
	int a = image[(y - 1)*width + (x - 1)] * 1 +
		image[(y - 1)*width + x] * 2 +
		image[(y - 1)*width + (x + 1)] * 1 +
		image[(y + 1)*width + (x - 1)] * (-1) +
		image[(y + 1)*width + x] * (-2) +
		image[(y + 1)*width + (x + 1)] * (-1);

	int b = image[(y - 1)*width + (x - 1)] * 1 +
		image[y*width + (x - 1)] * 2 +
		image[(y + 1)*width + (x - 1)] * 1 +
		image[(y - 1)*width + (x + 1)] * (-1) +
		image[y*width + (x + 1)] * (-2) +
		image[(y + 1)*width + (x + 1)] * (-1);

	int g = sqrt(float(a*a + b*b));

	g = g > 100 ? 255 : 0;
	f_image[y*width + x] = g;*/
	int g = 0;
	int index = 0;
	for (int m = y - 2; m<y + 3; m++)
	{
		for (int n = x - 2; n<x + 3; n++)
		{
			g += image[m*width + n] * templates[index++];
		}
	}
	g /= 273;
	if (g > 255)
		g = 255;
	f_image[id] = g;
}


int main()
{
	//double a = clock();
	Mat src;
	src = imread("D:\\testpic\\2.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	Mat dst = src.clone();
	for (int y = 0; y < src.rows; y++)
		for (int x = 0; x < src.cols; x++)
			dst.at<uchar>(y, x) = 0.0;
	/////////////////////////////////////////////
	for (int i = 0; i < src.rows; i++)
	{
		for (int j = 0; j < src.cols; j++)
		{
			image[i*src.cols + j] = src.at<uchar>(i, j);
		}
	}
	int image_size = src.rows*src.cols;
	int* cuda_image;
	int* cuda_f_image;
	double time_sum = 0;
	for (int h = 0; h < 10; h++)
	{
		double a = clock();
		hipMalloc((void**)&cuda_image, sizeof(int) * src.cols * src.rows);
		hipMalloc((void**)&cuda_f_image, sizeof(int) * src.cols * src.rows);
		hipMemcpy(cuda_image, image, sizeof(int) * src.rows * src.cols, hipMemcpyHostToDevice);
		gradient << <(image_size / 64) + 1, 64 >> > (cuda_image, cuda_f_image, src.rows, src.cols);
		hipMemcpy(f_image, cuda_f_image, sizeof(int) * src.rows * src.cols, hipMemcpyDeviceToHost);
		double b = clock();
		double diff = (b - a) / CLOCKS_PER_SEC;
		//cout << diff << endl;
		time_sum += diff;
	}
	cout << time_sum / 10 <<endl;


	for (int y = 0; y < src.rows ; y++) {
		for (int x = 0; x < src.cols ; x++) {
			dst.at<uchar>(y, x) = f_image[y*(src.cols) + x];
		}
	}

	cout << src.rows << endl;
	cout << src.cols << endl;

	namedWindow("initial", WINDOW_NORMAL);
	imshow("initial", src);

	namedWindow("final", WINDOW_NORMAL);
	imshow("final", dst);

	//double b = clock();
	//double diff = (b - a) / CLOCKS_PER_SEC;
	//cout << diff << endl;

	waitKey();
}
