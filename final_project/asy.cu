#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include <WINDOWS.H>
#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<time.h>

// includes, cuda
#include <hip/hip_vector_types.h>
#include <driver_functions.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// CUDA utilities and system includes
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <math.h>
using namespace std;
using namespace cv;

int image[200000000];
int f_image[200000000];

/*__global__ void gradient_x(int x, int y,int width)
{
int gx;
gx = image[(x - 1)*width + (y - 1)] * 1 +
image[(x - 1)*width + y] * 2 +
image[(x - 1)*width + (y + 1)] * 1 +
image[(x + 1)*width + (y - 1)] * (-1) +
image[(x + 1)*width + y] * (-2) +
image[(x + 1)*width + (y + 1)] * (-1);
}
__global__ void gradient_y(int x, int y,int width)
{
int gy;
gy = image[(x - 1)*width + (y - 1)] * 1 +
image[x*width + (y - 1)] * 2 +
image[(x + 1)*width + (y - 1)] * 1 +
image[(x - 1)*width + (y + 1)] * (-1) +
image[x*width + (y + 1)] * (-2) +
image[(x + 1)*width + (y + 1)] * (-1);
}*/

__global__ void gradient(int* image, int* f_image,int padding, int width, int length)
{
	//int a = gradient_x(x, y,width);
	//int b = gradient_y(x, y,width);
	int id = (blockIdx.x*blockDim.x + threadIdx.x)+padding;
	if (blockIdx.x*blockDim.x + threadIdx.x >= width)
	{
		return;
	}

	int y = id / width;
	int x = id % width;
	if (y == 0 || y == length - 1 || x == 0 || x == width - 1)
	{
		return;
	}
	int a = image[(y - 1)*width + (x - 1)] +
		image[(y - 1)*width + x] * 2 +
		image[(y - 1)*width + (x + 1)] -
		image[(y + 1)*width + (x - 1)] -
		image[(y + 1)*width + x] * 2 -
		image[(y + 1)*width + (x + 1)];

	int b = image[(y - 1)*width + (x - 1)] +
		image[y*width + (x - 1)] * 2 +
		image[(y + 1)*width + (x - 1)] -
		image[(y - 1)*width + (x + 1)] -
		image[y*width + (x + 1)] * 2 -
		image[(y + 1)*width + (x + 1)];

	int g = sqrt(float(a*a + b*b));

	g = g > 100 ? 255 : 0;
	f_image[id] = g;

}


int main()
{
	Mat src;
	src = imread("D:\\testpic\\666.png", CV_LOAD_IMAGE_GRAYSCALE);
	Mat dst = src.clone();
	for (int y = 0; y < src.rows; y++)
		for (int x = 0; x < src.cols; x++)
			dst.at<uchar>(y, x) = 0.0;
	/////////////////////////////////////////////
	for (int i = 0; i < src.rows; i++)
	{
		for (int j = 0; j < src.cols; j++)
		{
			image[i*src.cols + j] = src.at<uchar>(i, j);
		}
	}

	int* cuda_image;
	int* cuda_f_image;
	hipMalloc((void**)&cuda_image, sizeof(int) * src.rows * src.cols);
	hipMalloc((void**)&cuda_f_image, sizeof(int) * src.rows * src.cols);
	hipStream_t streams[32];
	for (int i = 0; i < 4; i++)
	{
		hipStreamCreate(&streams[i]);
	}
	double a = clock();
	hipMemcpyAsync(cuda_image + 0 * src.cols, image, sizeof(int)*src.cols, hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(cuda_image + 1 * src.cols, image + 1 * src.cols , sizeof(int)*src.cols, hipMemcpyHostToDevice, streams[0]);
	for (int i = 1; i < src.rows - 1; i++) {
		hipMemcpyAsync(cuda_image + (i + 1) * src.cols, image+(i + 1)*src.cols, sizeof(int) * src.cols, hipMemcpyHostToDevice, streams[i & 3]);
		 gradient<<<(src.cols + 127) >> 7, 128, 0, streams[i & 3] >> >(cuda_image, cuda_f_image, i * src.cols, src.cols, src.rows);
	}
	for (int i = 1; i < src.rows - 1; i++) {
		hipMemcpyAsync(f_image + i*src.cols, cuda_f_image + i * src.cols, sizeof(int) * src.cols, hipMemcpyDeviceToHost, streams[i & 3]);
	}
	for (int i = 0; i < 4; i++)
		hipStreamSynchronize(streams[i]);

	
	//hipMemcpy(cuda_image, image, sizeof(int) * src.rows * src.cols, hipMemcpyHostToDevice);

	//int image_size = src.rows*src.cols;
	//gradient <<<(image_size / 2048) + 1, 1024 >>> (cuda_image, cuda_f_image, src.rows, src.cols);

	//(f_image, cuda_f_image, sizeof(int) * src.rows * src.cols, hipMemcpyDeviceToHost);
	double b = clock();
	double diff = (b - a) / CLOCKS_PER_SEC;
	cout << diff << endl;



	for (int y = 1; y < src.rows - 1; y++) {
		for (int x = 1; x < src.cols - 1; x++) {
			dst.at<uchar>(y, x) = f_image[y*(src.cols) + x];
		}
	}

	cout << src.rows << endl;
	cout << src.cols << endl;

	namedWindow("initial", WINDOW_NORMAL);
	imshow("initial", src);

	namedWindow("final", WINDOW_NORMAL);
	imshow("final", dst);



	waitKey();
}
