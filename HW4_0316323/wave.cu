
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
__global__ void init_line(float*, float*, int);
__global__ void update (float*, float*, int, int);
void printfinal (void);

int nsteps,tpoints,rcode;               
int alloc_points;
float* values,*oldval,*newval;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ void init_line(float* _oldval,float* _newval,int _tpoints)
{
   float x;
   
   int i = blockIdx.x*blockDim.x+threadIdx.x;
   if(i<_tpoints)
   {
      x = (float)i/(_tpoints - 1);
      _newval[i] = __sinf(6.2831853 * x);
      _oldval[i] = _newval[i];
   }
}

__global__ void update(float* _oldval,float* _newval,int _tpoints,int _nsteps)
{
   //int i, j;

   int i = blockIdx.x*blockDim.x+threadIdx.x;
   if(i<_tpoints)
   {
      float local_oldval = _oldval[i];
      float local_newval = _newval[i];
      float local_nextval;
      
      for(int j=0;j<_nsteps;j++)
      {
         if( (i==0) || (i==_tpoints-1))
         {
            local_nextval = 0;
         }
         else 
         {
            local_nextval = 1.82*local_newval - local_oldval;
         }
         local_oldval = local_newval;
         local_newval = local_nextval;
      }
      _newval[i] = local_newval;
   }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 0; i < tpoints; i++) {
      printf("%6.4f ", values[i]);
      if ((i+1)%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();

   alloc_points = tpoints + 255;
   values = (float*) malloc(alloc_points * sizeof(float));
   if(!values)
   {
      exit(EXIT_FAILURE);
   }

   hipMalloc((void**) &oldval , alloc_points*sizeof(float));
   hipMalloc((void**) &newval , alloc_points*sizeof(float));

   dim3 threadsPerBlock(256);
   dim3 numOfBlocks(alloc_points/256);



	printf("Initializing points on the line...\n");
	init_line<<<numOfBlocks,threadsPerBlock>>>(oldval,newval,tpoints);

	printf("Updating all points for all time steps...\n");
	update<<<numOfBlocks,threadsPerBlock>>>(oldval,newval,tpoints,nsteps);
	printf("Printing final results...\n");
   hipMemcpy(values, newval, alloc_points * sizeof(float), hipMemcpyDeviceToHost);
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
